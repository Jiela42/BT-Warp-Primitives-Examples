#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <iostream>

using namespace std;




__global__ void elmt_wise_mult(float * a, float * b, float * res){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    res[id] = a[id] * b[id];
}

void init_Ones(float* a, int size){
    for(int i = 0; i < size; i++){
        a[i] = 1;
    }
}


int main(){

    int size = 1 << 6;

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float));

    float *d_a, *d_b, *d_res, *d_temp;

    hipMalloc(&d_a, sizeof(float) * size);
    hipMalloc(&d_b, sizeof(float) * size);
    hipMalloc(&d_res, sizeof(float));
    hipMalloc(&d_temp, sizeof(float)*size);

    init_Ones(a, size);
    init_Ones(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    elmt_wise_mult<<<1,size>>>(d_a, d_b, d_temp);

    hipDeviceSynchronize();

    void *d_temp_storage= NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_temp, d_res, size);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_temp, d_res, size);

    hipMemcpy(res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    cout << "Result: " << *res << "\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);

    return 0;
}
