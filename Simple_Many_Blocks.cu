
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>

// try to do vector ops of sum of a*b (where a and b are vectors) So basically the inner product

// This is a function, that can be useful for debugging
__global__ void printArr(float * a){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    printf("From PrintArr: res[%d] = %f\n", id, a[id]);
}

__global__ void copy(float * a, float* res){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    // if(id == 0){
    //     printf( "Copy has been entered\n");
    // }
    // printf("I will put a[%d]= %f into res[%d]\n", id, a[id], id);
    res[id] = a[id];
}

__global__ void align (float* a, float* res, int targetSize, int stride){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    res[id] = a[id * stride];
    // printf("I put a[%d] = %f into res[%d]\n", id* stride, a[id*stride], id);
}

__global__ void naiveGlobalMem(float * a, float * b, float* res, int size){
   
    /* since every thread adds up two numbers (in the first iteration)
    we need double the block-starting point to keep working on disjoint parts of the input */
    int id = threadIdx.x + 2*(blockIdx.x * blockDim.x);
    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    // first iteration: 
    res[id] = (a[id] * b[id]) + (a[id + stepSize] * b[id + stepSize]);
    stepSize /= 2; 

    for(int i = 1; i < iterations; i++){
        if (id - 2 * (blockDim.x * blockIdx.x) < stepSize){
            if(id == 112 || id == 108){
                //printf("I am thread %d, with res[%d] = %f. I will add res[%d] = %f\n", id, id, res[id], id + stepSize, res[id + stepSize]);
            }
            res[id] += res[id + stepSize];
            stepSize /= 2;
        }
        __syncthreads();
    }
}

__global__ void globalMemSum(float * res, int size){
    int id = threadIdx.x + 2 * (blockDim.x * blockIdx.x);
    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    if(id == 0){
        // printf("globalMemSum has been invoked\n");
    }
    for(int i = 0; i < iterations; i++){
        if ((id - 2 * (blockDim.x * blockIdx.x)) < stepSize){
            //printf("I am thread %d, with res[%d] = %f. I will add res[%d] = %f\n", id, id, res[id], id + stepSize, res[id + stepSize]);   
            res[id] += res[id + stepSize];
            stepSize /= 2;
        }
        __syncthreads();
    }
    if(id == 0){
        //printf("the result is res[0] = %f\n", res[0]);
    }
}

__global__ void naiveSharedMem(float * a, float * b, float* res, int size){

    int bSize = blockDim.x;
    extern __shared__ float rab[];
    
    // because we are computing this result blockwise "local" aka in shared memory,
    // the global vs. the blockId are different
    int gId = threadIdx.x + 2 * (blockDim.x * blockIdx.x);
    int bId = threadIdx.x;
    int stepSize = size/2;
    
    int s_a_i = bId + size;
    int s_b_i = bId + (2 * size);

    if(gId == 0){
        printf("Entered kernel\n");
    }
    
    //loading into shared Memory
    rab[s_a_i] = a[bId];
    rab[s_b_i] = b[bId];
    rab[s_a_i + stepSize] = a[bId + stepSize];
    rab[s_b_i + stepSize] = b[bId + stepSize];

    // First Iteration
    rab[bId] = rab[s_a_i] * rab[s_b_i] + (rab[s_a_i + stepSize] * rab[s_b_i + stepSize]);

    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    for(int i = 1; i < iterations; i++){
        if (bId < stepSize){
            // printf("I am Tread %d, this is iteration %d, and I will add cell %d\n", id, i, id+ stepSize);
            rab[bId] += rab[bId + stepSize];
            stepSize /= 2;
        }
        __syncthreads();
    }

    if (bId == 0){
        res[gId] = rab[bId];
    }
}

__global__ void sharedMemSum (float * res, int size){

}

__global__ void naiveWarpRed(float* a, float* b, float* res, int size){

    int id = threadIdx.x;
    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    float mySum = a[id] * b [id];

    for(int i = 0; i < iterations-1; i++){
        int condition = id < stepSize;
        unsigned mask = __ballot_sync(0xffffffff, condition);
        if(condition){
            mySum += __shfl_up_sync(mask, mySum, stepSize);
            stepSize /= 2;
        }
    }
    if(id == 0){
        res[id] = mySum;
    }
}

void vecInitOnes(float * a, int size){

    for (int i = 0; i < size; i++){
        a[i] = 1.0;
    }
}

void vecInitGauss (float * a, int size){
    for (int i = 0; i < size; i++){
        a[i] = i;
    }
}

void vecInit(float * a, float size){
    srand(time(NULL));

    for(int i = 0; i < size; i++){
        float r = rand() % 64;
        a[i] = r;
    }
}

void callCublas(){
    hipblasHandle_t h;
    hipblasCreate(&h);

    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    int size = 32;
    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    hipblasSdot(h, size, d_a, 1, d_b, 1, d_res);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(h);

    printf("Calling Cublas took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for(int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveGlobalMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    // Every thread can add two numbers, thus we can add 256 number in one iteration
    int bSize = size / (2 * threads);

    float n = size - 1.0;
    float expectedRes = (pow(n, 2.0) + n) / 2;

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // Random Vector initializsation:
    // vecInit (a, size);
    // vecInit (b, size);

    // vecInitGauss(a, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // to support a size bigger than 32 we need to
    // basically do another addition for each one of the blocks!

    //LOOOOOOOOL this only applies to standard warp reduction, not using global or shared Memory!!

    // Every thread can add two numbers, thus we can add 2*threads many numbers in one iteration
    naiveGlobalMem <<<bSize, threads>>> (d_a, d_b, d_res, threads * 2);
    
    
    // bSize is the number of elements that still need summing up
    while (bSize > threads){
        
        hipDeviceSynchronize();
        int newBSize = bSize / (threads * 2);
        align <<<2*newBSize, threads>>> (d_res, d_a, bSize, 2*threads);
        hipError_t err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error in align: %s\n", hipGetErrorString(err));
            printf("Config Args: newBsize = %d, threads = %d\n", newBSize, threads);
        }
        hipDeviceSynchronize();

        // Align swaps around d_a and d_res, so we swap it back
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;

        bSize = newBSize;
        globalMemSum <<<bSize, threads>>> (d_res, threads * 2);
    }
    
    if (bSize > 1){
        // printf("Entered bSize > 1 with bSize = %d\n", bSize);
        // printf("After alignment:\n");
        // printArr <<<1, bSize>>> (d_res);
        hipDeviceSynchronize();
        align <<<1, bSize>>> (d_res, d_a, bSize, 2*threads);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error in align: %s\n", hipGetErrorString(err));
            printf("Config Args: newBsize = %d, threads = %d\n", 1, bSize);
        }

        // Align swaps around d_a and d_res, so we swap it back
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;

        globalMemSum <<<1, bSize>>> (d_res, bSize);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Naive with Global Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveSharedMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    int bSize = size / (2 * threads);
    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);


    // There are 2 * threads elements to be added and there are three arrays of that many elements
    naiveSharedMem<<<bSize, threads, sizeof(float) * threads * 2 * 3>>> (d_a, d_b, d_res, threads * 2);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        printf("Cuda Error before printArr: %s\n", hipGetErrorString(err));
        printf("Launch config: bsize = %d, threads = %d, sharedMem = %d\n", bSize, threads, threads * 2 * 3);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Naive with Shared Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for(int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveWarpRed(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    naiveWarpRed<<<1,size>>> (d_a, d_b, d_res, size);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("WarpReduction Version took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for (int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

int main(){


    int size = 1 << 16;
    int threads = 1 << 5;

    printf("Threads = %d, size = %d\n", threads, size );

    for (int i = 0; i < 5; i++){
    }
    
    // callCublas();
    callNaiveGlobalMem(size, threads);
    callNaiveSharedMem(size, threads);
   // callNaiveWarpRed(size, threads); 

    return 0;
}
