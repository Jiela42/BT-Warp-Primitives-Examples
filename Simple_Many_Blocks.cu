
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>

// try to do vector ops of sum of a*b (where a and b are vectors) So basically the inner product

// This is a function, that can be useful for debugging
__global__ void printArr(float * a){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    printf("From PrintArr: res[%d] = %f\n", id, a[id]);
}

//another function useful for debugging:
__global__ void printVal(float * a){
    printf("The requested Value is %f\n", a[threadIdx.x]);
}

__global__ void copy(float * a, float* res){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    // if(id == 0){
    //     printf( "Copy has been entered\n");
    // }
    // printf("I will put a[%d]= %f into res[%d]\n", id, a[id], id);
    res[id] = a[id];
}

//elaborated function to check which block and how is not doing alirght in Gaussian
__global__ void gaussCheck(float * a){

    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    int low;
    id == 0 ? low = 0 : low = 4095 + (id - 1) * 4096;
    int top = 4095 + id * 4096;

    float subGauss = ((pow(top, 2.0) + top) / 2) - ((pow(low, 2.0) + low) / 2);
    if (blockDim.x == 4){
        printf("I am Thread %d, my subGauss is %f\n", id, subGauss);
    }
    if (subGauss =! a[id]){
        printf("In the previous iteration block %d failed, produced %f, should be %f\n", id, a[id], subGauss);
    }


}

__global__ void align (float* a, float* res, int stride){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    res[id] = a[id * stride];
    //printf("I put a[%d] = %f into res[%d], the stride is %d\n", id* stride, a[id*stride], id, stride);
}

__global__ void naiveGlobalMem(float * a, float * b, float* res, int size){
   
    /* since every thread adds up two numbers (in the first iteration)
    we need double the block-starting point to keep working on disjoint parts of the input */
    int id = threadIdx.x + 2*(blockIdx.x * blockDim.x);
    int stepSize = size/2;

    // first iteration: 
    res[id] = (a[id] * b[id]) + (a[id + stepSize] * b[id + stepSize]);

    __syncthreads();

    for(int i = stepSize / 2; i > 0; i /= 2){
        if (id - 2 * (blockDim.x * blockIdx.x) < i){
            if(id == 112 || id == 108){
                //printf("I am thread %d, with res[%d] = %f. I will add res[%d] = %f\n", id, id, res[id], id + stepSize, res[id + stepSize]);
            }
            res[id] += res[id + i];
            stepSize /= 2;
        }
        __syncthreads();
    }
}

__global__ void globalMemSum(float * res, int size){
    int id = threadIdx.x + 2 * (blockDim.x * blockIdx.x);

    for(int i = size / 2; i > 0; i /= 2){
        if ((id - 2 * (blockDim.x * blockIdx.x)) < i){
            //printf("I am thread %d, with res[%d] = %f. I will add res[%d] = %f\n", id, id, res[id], id + stepSize, res[id + stepSize]);   
            res[id] += res[id + i];
        }
        __syncthreads();
    }
    if(id == 0){
        //printf("the result is res[0] = %f\n", res[0]);
    }
}

__global__ void naiveSharedMem(float * a, float * b, float* res, int size){

    extern __shared__ float rab[];
    
    // because we are computing this result blockwise "local" aka in shared memory,
    // the global vs. the blockId are different
    int bId = threadIdx.x;
    int gId = threadIdx.x + 2 * (blockDim.x * blockIdx.x);
    int stepSize = size / 2;
    
    int s_a_i = bId + size;
    int s_b_i = bId + (2 * size);

    //loading into shared Memory
    rab[s_a_i] = a[gId];
    rab[s_b_i] = b[gId];
    rab[s_a_i + stepSize] = a[gId + stepSize];
    rab[s_b_i + stepSize] = b[gId + stepSize];

   
    // First Iteration
    rab[bId] = rab[s_a_i] * rab[s_b_i] + (rab[s_a_i + stepSize] * rab[s_b_i + stepSize]);
    //this is just a variable for debugging purposes
    int it = 2;
    __syncthreads();
    for(int i = stepSize / 2; i > 0; i /= 2){
        if (bId < i){
            rab[bId] += rab[bId + i];
            it ++;
        }
        __syncthreads();
    }

    // Writing the result bacck to global Memory
    if (bId == 0){
      //  printf("I am from block %d, I will write %f into cell %d\n", blockIdx.x, rab[bId], gId);
        res[gId] = rab[bId];
    }
}

__global__ void sharedMemSum (float * res, int size){

    extern __shared__ float r[];
    int bId = threadIdx.x;
    int gId = threadIdx.x + 2 * (blockDim.x * blockIdx.x);

    // if(bId== 0){

    //     printf("Blocksize is %d, size is %d\n", blockDim.x,size);
    // }

    // First iteration + loading:
    r[bId] = res[gId] + res[gId + size / 2];
    
    if(gridDim.x == 1){

        printf("I am thread %d, and after the first iteration my value is %f\n", bId, r[bId]);
    }

    __syncthreads();
    int its = 0;
    for(int i = size / 4; i > 0; i /= 2){
        
        if(bId < i){

            r[bId] += r[bId + i];
            if(gridDim.x == 1){
               printf("I am thread %d and after iteration %d my value is %f\n", bId, its, r[bId]);
            }
        }
        __syncthreads();
        its++;
    }

    // writing the result back to global Memory
    if(bId == 0){
        if(gridDim.x == 1){
         printf("I am thread 0 from block %d and my result is %f, my gid is %d\n", blockIdx.x, r[bId], gId);
        }
        res[gId] = r[bId];
    }
    
}

__global__ void naiveWarpRed(float* a, float* b, float* res, int size){

    extern __shared__ float r[];

    int id = threadIdx.x + 2 * (blockDim.x * blockIdx.x);
    int blockId = threadIdx.x;
    int laneId = threadIdx.x % 32;
    int warpId = threadIdx.x / 32;
    int nWarps = blockDim.x / 32;
    int stepSize = size / 2;

    // Loading and first iteration:
    float mySum = a[id] * b[id] + a[id + stepSize] * b[id + stepSize];

    // First warp Reduction
    for(int i = min (size / 4, 16); i > 0; i /= 2){

        // The threads that values need to be read from, must partition in the shuffle!
        int condition = laneId < (i * 2);
        unsigned mask = __ballot_sync(0xffffffff, condition);

        if(condition){
            if(blockIdx.x == 1){
                // printf("I am thread %d, mySum is %f before addition, i is %d\n", id, mySum, i);
            }
            mySum += __shfl_down_sync(mask, mySum, i);

            if(blockIdx.x == 1){
                // printf("I am thread %d, mySum is %f after addition\n", id, mySum);
            }
        }
    }

    // loading warp results into shared Memory
    if (laneId == 0){
        // printf("from first kernel call: I am thread %d, in lane 0 of warp %d and mySum is %f\n", id, warpId, mySum);
        // printf("from first kernel call: nWarps is %d\n", nWarps);
        r[warpId] = mySum;
    }

    __syncthreads();
    // Reducing results from first reduction
    if (nWarps > 1){
        for (int i = nWarps / 2; i > 0; i /= 2){
            if(blockId < i){
                //printf("from first kernel call: I am thread 0 from block %d, I will add %f to %f\n", blockId, r[blockId], r[blockId + i]);
                r[blockId] += r[blockId + i];
            }
            __syncthreads();
        }
    }

    // Writing back to global Memory
    if(threadIdx.x == 0){
     //   printf("Hit in writeback in first call, I am thread %d and I am writing back %f\n", id, mySum);
        res[blockIdx.x] = r[0];
    }
}
__global__ void warpRedSum(float* a, float* res, int size){

    extern __shared__ float r[];

    int id = threadIdx.x + (blockDim.x * blockIdx.x);
    int blockId = threadIdx.x;
    int laneId = threadIdx.x % 32;
    int warpId = threadIdx.x / 32;
    int nWarps = blockDim.x / 32;
    int stepSize = size / 2;

    // First Iteration
    float mySum = a[id] + a[id + stepSize];


    // First warp Reduction
    // The number of iterations is adjusted to account for the possiblity of size == 2
    for(int i = min(16, size / 4); i > 0; i /= 2){
        // The threads that values need to be read from, must partition in the shuffle!
        int condition = laneId < (i * 2);
        unsigned mask = __ballot_sync(0xffffffff, condition);

        if(condition){
            mySum += __shfl_down_sync(mask, mySum, i);
        }
    }

    // loading warp results into shared Memory
    if (laneId == 0){
        // printf("I am thread %d, in lane 0 of warp %d and mySum is %f\n", id, warpId, mySum);
        // printf("nWarps is %d\n", nWarps);
        r[warpId] = mySum;
    }

    __syncthreads();
    // Reducing results from first reduction
    if (nWarps > 1){
        for (int i = nWarps / 2; i > 0; i /= 2){
            if(blockId < i){
                // printf("I am thread 0 from block %d, I will add %f to %f\n", blockId, r[blockId], r[blockId + i]);
                r[blockId] += r[blockId + i];
            }
            __syncthreads();
        }
    }

    // Writing back to global Memory
    if(threadIdx.x == 0){
      //printf("Hit in writeback, I am thread %d and I am writing back %f\n", id, mySum);
        res[blockIdx.x] = r[0];
    }
}

void vecInitOnes(float * a, int size){

    for (int i = 0; i < size; i++){
        a[i] = 1.0;
    }
}

void vecInitGauss (float * a, int size){
    for (int i = 0; i < size; i++){
        a[i] = i;
    }
}

void vecInit(float * a, float size){
    srand(time(NULL));

    for(int i = 0; i < size; i++){
        float r = rand() % 64;
        a[i] = r;
    }
}

void testSharedMemSum(int size, int threads){

    int nBlocks = size / (2 * threads);
    float * a = (float*) malloc (sizeof(float)*size);
    float* d_a;

    hipMalloc(&d_a, sizeof(float)*size);

    vecInitGauss (a, size);
    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);

    sharedMemSum <<<nBlocks, threads, sizeof(float) * threads>>> (d_a, size);

    hipMemcpy(a, d_a, sizeof(float)*size, hipMemcpyDeviceToHost);

    float n = size - 1;
    float expectedRes = (pow(n, 2.0) + n) / 2;
    if (a[0] != (float) expectedRes){
        printf("The result is %f, but should be %f \n", a[0], (float) expectedRes);
    }
}

float subGauss(int low, int top){
    
    return (((pow(top, 2.0) + top) / 2) - ((pow(low, 2.0) + low) / 2));
}

void callCublas(){
    hipblasHandle_t h;
    hipblasCreate(&h);

    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    int size = 32;
    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    hipblasSdot(h, size, d_a, 1, d_b, 1, d_res);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(h);

    printf("Calling Cublas took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for(int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveGlobalMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    // Every thread can add two numbers, thus we can add 256 number in one iteration
    int nBlocks = size / (2 * threads);

    float n = size - 1.0;
    float expectedRes = (pow(n, 2.0) + n) / 2;

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // Random Vector initializsation:
    // vecInit (a, size);
    // vecInit (b, size);

   vecInitGauss(a, size);

    //vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // to support a size bigger than 32 we need to
    // basically do another addition for each one of the blocks!

    //LOOOOOOOOL this only applies to standard warp reduction, not using global or shared Memory!!

    // Every thread can add two numbers, thus we can add 2*threads many numbers in one iteration
    naiveGlobalMem <<<nBlocks, threads>>> (d_a, d_b, d_res, threads * 2);
    
    
    // nBlocks is the number of elements that still need summing up
    while (nBlocks > threads){
        //careful! There is no case that handles if the number of elements to add are somewhere inbetween threads and 2*threads
        hipDeviceSynchronize();
        int new_nBlocks = nBlocks / (threads * 2);
        align <<<2 * new_nBlocks, threads>>> (d_res, d_a, 2 * threads);
        hipError_t err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error in align: %s\n", hipGetErrorString(err));
            printf("Config Args: new_nBlocks = %d, threads = %d\n", new_nBlocks, threads);
        }
        hipDeviceSynchronize();

        // Align swaps around d_a and d_res, so we swap it back
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;

        nBlocks = new_nBlocks;
        globalMemSum <<<nBlocks, threads>>> (d_res, threads * 2);
    }
    
    if (nBlocks > 1){
        // printf("Entered nBlocks > 1 with nBlocks = %d\n", nBlocks);
        // printf("After alignment:\n");
        // printArr <<<1, nBlocks>>> (d_res);
        hipDeviceSynchronize();
        align <<<1, nBlocks>>> (d_res, d_a, 2 * threads);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error in align: %s\n", hipGetErrorString(err));
            printf("Config Args: new_nBlocks = %d, threads = %d\n", 1, nBlocks);
        }

        // Align swaps around d_a and d_res, so we swap it back
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;

        globalMemSum <<<1, nBlocks / 2>>> (d_res, nBlocks);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // printf("Naive with Global Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);
    
    if (res[0] != (float)expectedRes){
        printf("The result is %f, but should be %f \n", res[0], (float) expectedRes);
    } else {
        printf("Result is correct\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveSharedMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    int nBlocks = size / (2 * threads);
    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitGauss (a, size);

    //vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    // printf("This is before first iteration: nBlocks = %d\n", nBlocks);

    // There are 2 * threads elements to be added and there are three arrays of that many elements
    naiveSharedMem<<<nBlocks, threads, sizeof(float) * threads * 2 * 3>>> (d_a, d_b, d_res, threads * 2);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        printf("Cuda Error after first iteration: %s\n", hipGetErrorString(err));
        printf("Launch config: nBlocks = %d, threads = %d, sharedMem = %d\n", nBlocks, threads, threads * 2 * 3);
    }
    
    int new_nBlocks = nBlocks / (threads * 2);
    if (nBlocks > 1 && new_nBlocks == 0){
        new_nBlocks = 1;
    }
    
    hipDeviceSynchronize();
    
    err = hipGetLastError();
    if(err != hipSuccess){
        printf("Cuda Error after align: %s\n", hipGetErrorString(err));
        printf("Launch config: new_nBlocks = %d, threads = %d, sharedMem = %d\n", new_nBlocks, threads, threads * 2 * 3);
    }
    int its = 0;
    // printf("nBlocks before while: %d\n",nBlocks);
    while(nBlocks > 2 * threads){
        its++;
        //printf("This is while iteration %d, new_nBlocks = %d, nBlocks = %d\n", its, new_nBlocks, nBlocks);
        
        align<<<new_nBlocks * 2, threads>>> (d_res, d_a, 2 * threads);
        
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;
        
        hipDeviceSynchronize();
        gaussCheck <<<new_nBlocks, threads>>> (d_res);
        
       // printArr<<<new_nBlocks, threads>>> (d_res);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error after Align in while: %s\n", hipGetErrorString(err));
            printf("Launch config: #blocks = %d, threads = %d\n", new_nBlocks * 2, threads);
        }
        
        nBlocks = new_nBlocks;
        sharedMemSum<<<nBlocks, threads, sizeof(float) * threads>>> (d_res, threads * 2);

        hipDeviceSynchronize();

        err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error after SharedMemSum in while: %s\n", hipGetErrorString(err));
            printf("Launch config: nBlocks = %d, threads = %d, sharedMem = %d\n", nBlocks, threads, threads);
        }
        new_nBlocks = nBlocks / (threads * 2);
        if (nBlocks > 1 && new_nBlocks == 0){
            new_nBlocks = 1;
        }
    }

    if (nBlocks > 1){
        align<<<new_nBlocks, nBlocks>>> (d_res, d_a, 2 * threads);
        
        //  printf("In if new_nBlocks = %d, nBlocks is %d\n", new_nBlocks, nBlocks);
        
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;
        
        hipDeviceSynchronize();
        
        err = hipGetLastError();
         if(err != hipSuccess){
             printf("Cuda Error after align in if: %s\n", hipGetErrorString(err));
             printf("Launch config: new_nBlocks = %d, threads = %d\n", new_nBlocks, nBlocks);
        }
        // printf("from if clause: nBlocks= %d\n", nBlocks);
         printArr <<<1, nBlocks>>> (d_res);


        gaussCheck <<<1, nBlocks>>> (d_res);

        sharedMemSum<<<1, (nBlocks / 2), sizeof(float) * (nBlocks / 2)>>> (d_res, nBlocks);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //printf("Naive with Shared Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);
    
    float expectedRes = subGauss(0, size - 1);

    if (res[0] != (float) expectedRes){
        printf("Threads = %d, size = %d \nThe result is %f, but should be %f \n", threads, size, res[0], (float) expectedRes);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveWarpRed(int size, int threads){
    hipEvent_t start, stop;
    float time;
    
    int nBlocks = size / (2 * threads);
    int warps_per_block = threads / 32 + 1;     // the +1 is to ensure we assign memory even if int division rounds down

    hipEventCreate (&start);
    hipEventCreate(&stop);

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitGauss(a, size);

   // vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    naiveWarpRed<<<nBlocks, threads, sizeof(float) * warps_per_block>>> (d_a, d_b, d_res, threads * 2);
    hipDeviceSynchronize();
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        printf("Cuda Error after first iteration: %s\n", hipGetErrorString(err));
        printf("Launch config: gridDim = %d, threads = %d, sharedMem = %d\n", 1, size / 2, warps_per_block);
    }
    int new_nBlocks = nBlocks / (threads * 2);

    //printVal<<<1,1>>>(d_res);
    hipDeviceSynchronize();

    while (nBlocks > threads * 2){

        nBlocks = new_nBlocks;
        warpRedSum<<<nBlocks, threads, sizeof(float) * warps_per_block>>>(d_res, d_a, threads * 2);
        hipDeviceSynchronize();
        //printVal<<<1,1>>>(d_res);

        float * temp = d_res;
        d_res = d_a;
        d_a = temp;


        new_nBlocks = nBlocks / (threads * 2);
     //   printf("In While:\n");
       // printArr<<<new_nBlocks * 2, threads>>>(d_res);
    }

   // printf("Between While and if\n");
    //printArr<<<new_nBlocks * 2, threads>>>(d_res);
    //printVal<<<1,1>>>(d_res);

    if (nBlocks > 1){
        
        nBlocks /= 2;
        warps_per_block = nBlocks / 32 + 1;         // the plus 1 is in case integer summation rounds down

        warpRedSum<<<1, nBlocks, sizeof(float) * warps_per_block>>> (d_res, d_a, nBlocks * 2);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if(err != hipSuccess){
            printf("Cuda Error after first iteration: %s\n", hipGetErrorString(err));
            printf("Launch config: gridDim = %d, threads = %d, sharedMem = %d\n", 1, nBlocks, warps_per_block);
    }

        float * temp = d_res;
        d_res = d_a;
        d_a = temp;        
    }


    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    float expRes = subGauss(0, size-1);
    
    if (res[0] != (float)expRes){
        printf("In Warp Reduction, the result is %f, but should be %f with %d threads and size %d \n", res[0], (float) expRes, threads, size);
    } else {
        printf("WarpReduction Version with %d Threads and Size %d took %fms\n",threads, size, time);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

int main(){

    for (int it = 0; it < 1; it ++){
        
        for (int i = 0; i <= 10; i ++){
            for (int j = i + 1; j < 27; j++){
                
                int size = 1 << j;
                int threads = 1 << i;
                //printf("Threads = %d, size = %d\n", threads, size );
                // callNaiveGlobalMem(size, threads);
                // callNaiveSharedMem(size, threads);
                callNaiveWarpRed(size, threads);
       
            }
        }
    }

    int j = 6;
    int i = 5;


    int size = 1 << j;
    int threads = 1 << i;
    
    
    // callCublas();
    //callNaiveGlobalMem(size, threads);
    // callNaiveSharedMem(size, threads);
   // callNaiveWarpRed(size, threads); 
   //testSharedMemSum(2048, 1024);

    return 0;
}
