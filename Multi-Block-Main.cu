#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>
#include "multi_block_globalMem.h"
#include "multi_block_sharedMem.h"
#include "multi_block_warps.h"

#define cudaCheckErr() {                                                                           \
    hipError_t err = hipGetLastError();                                                          \
    if(err != hipSuccess){                                                                        \
    printf("Cuda Error in align: %s, %d %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
    exit(0);                                                                                   \
    }                                                                                              \
}

// do vector ops of sum of a*b (where a and b are vectors) So basically the inner product

void runTest(char* lab, int (*fptr)(float* a, float* b, float* res, int size, int threads), float* src_a, float* src_b, float* src_res, int size, int threads, int reps, float checkSum){

    float *d_a, *d_b, *d_res;

    hipMalloc(&d_a, sizeof(float) * size * reps);
    hipMalloc(&d_b, sizeof(float) * size * reps);
    hipMalloc(&d_res, sizeof(float) * size * reps);

    hipEvent_t start, stop;
    float time_ms;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    cudaCheckErr();

    // creates the input reps times
    for (int i = 0; i < reps; i ++){
        int offset = size * i;
        hipMemcpy(d_a + offset, src_a, sizeof(float)*size , hipMemcpyHostToDevice);
        hipMemcpy(d_b + offset, src_b, sizeof(float)*size , hipMemcpyHostToDevice);
    }
    // warm up
    // fptr(d_a, d_b, d_res, size, threads);

    hipDeviceSynchronize();
    cudaCheckErr();

    hipEventRecord(start);
    int swap = 0;
    for(int i = 0; i < reps; i ++){
        int offset = size * i;
        swap = fptr(d_a + offset, d_b + offset, d_res, size, threads);
    }

    hipEventRecord(stop);
    hipDeviceSynchronize();
    cudaCheckErr();

    hipEventElapsedTime(&time_ms, start, stop);
    float time_s = time_ms / (float) 1e3;

    float GB = (float) size * sizeof(float) * reps * 2;
    float GBs = GB / time_s / (float)1e9;

    if(swap){
        float * temp = d_res;
        d_res = d_a;
        d_a = temp;
    }

    hipMemcpy(src_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErr();

    if (src_res[0] != checkSum){
        printf("%s with %d threads and size %d: result is %f, but should be %f \n",lab, threads, size, src_res[0], (float) checkSum);
    } else {
        printf("%s with %d threads and size %d TIME: %fs GB/s: %f\n", lab, threads, size, time_s, GBs);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
}


void vecInitOnes(float * a, int size){

    for (int i = 0; i < size; i++){
        a[i] = 1.0;
    }
}

void vecInitGauss (float * a, int size){
    for (int i = 0; i < size; i++){
        a[i] = i;
    }
}

int main(){

    int reps = 1;

    for (int i = 9; i < 10; i ++){         // Threads
        for (int j = 24; j < 25; j++){   // Number of Elements
            
            int size = 1 << j;
            int threads = 1 << i;

            float checkSum = ((size - 1) * (size - 1) + (size - 1)) / 2;
            checkSum = size;


            // Initialize a, b and the result in "normal" memory
            float * a = (float*) malloc (sizeof(float)*size);
            float * b = (float*) malloc (sizeof(float)*size);
            float * res = (float*) malloc (sizeof(float) * size);

            vecInitOnes(a, size);
            vecInitOnes(b, size);

            // callNaiveGlobalMem(size, threads);
            // runTest("multi_block_globalMem", multi_block_globalMem, a, b, res, size, threads, reps, checkSum);
            // callNaiveSharedMem(size, threads);
            // runTest("multi_block_SharedMem", multi_block_sharedMem, a, b, res, size, threads, reps, checkSum);
            
            // callNaiveWarpRed(size, threads);
            runTest("multi_block_Warps", multi_block_warps, a, b, res, size, threads, reps, checkSum);
            

            free(a);
            free(b);
            free(res);
        }
    }
}



