
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>

// try to do vector ops of sum of a*b (where a and b are vectors) So basically the inner product

__global__ void printArr(float * a){
    // This is just a function, to help debug
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    printf("From PrintArr: res[%d] = %f\n", id, a[id]);
}

__global__ void align (float* a, float* res, int targetSize, int stride){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    res[id] = a[id * stride];
    // printf("I put a[%d] = %f into res[%d]\n", id* stride, a[id*stride], id);
}

__global__ void naiveGlobalMem(float * a, float * b, float* res, int size){
   
    /* since every thread adds up two numbers (in the first iteration)
    we need double the block-starting point to keep working on disjoint parts of the input */
    int id = threadIdx.x + 2*(blockIdx.x * blockDim.x);
    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    // first iteration: 
    res[id] = (a[id] * b[id]) + (a[id + stepSize] * b[id + stepSize]);
    stepSize /= 2; 

    for(int i = 1; i < iterations; i++){
        if (id - 2 * (blockDim.x * blockIdx.x) < stepSize){
            if(id == 112 || id == 108){
                //printf("I am thread %d, with res[%d] = %f. I will add res[%d] = %f\n", id, id, res[id], id + stepSize, res[id + stepSize]);
            }
            res[id] += res[id + stepSize];
            stepSize /= 2;
        }
        __syncthreads();
    }
}

__global__ void naiveSharedMem(float * a, float * b, float* res, int size){
    __shared__ float rab[3 * 32 * 32];

    int id = threadIdx.x;
    
    int s_a_i = id + 32 * 32;
    int s_b_i = id + (2 * 32 * 32);
    
    rab[s_a_i] = a[id];
    rab[s_b_i] = b[id];

    rab[id] = rab[s_a_i] * rab[s_b_i];

    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    for(int i = 0; i < iterations; i++){
        if (id <stepSize){
            // printf("I am Tread %d, this is iteration %d, and I will add cell %d\n", id, i, id+ stepSize);
            rab[id] += rab[id + stepSize];
            stepSize /= 2;
        }
        __syncthreads();
    }

    if (id == 0){
        res[id] = rab[id];
    }
}

__global__ void naiveWarpRed(float* a, float* b, float* res, int size){

    int id = threadIdx.x;
    int stepSize = size/2;
    
    int iterations = 1;
    int logSize = size;

    while(logSize > 1){
        logSize /= 2;
        iterations++;
    }

    float mySum = a[id] * b [id];

    for(int i = 0; i < iterations-1; i++){
        int condition = id < stepSize;
        unsigned mask = __ballot_sync(0xffffffff, condition);
        if(condition){
            mySum += __shfl_up_sync(mask, mySum, stepSize);
            stepSize /= 2;
        }
    }
    if(id == 0){
        res[id] = mySum;
    }
}

void vecInitOnes(float * a, int size){

    for (int i = 0; i < size; i++){
        a[i] = 1.0;
    }
}

void vecInitGauss (float * a, int size){
    for (int i = 0; i < size; i++){
        a[i] = i;
    }
}

void vecInit(float * a, float size){
    srand(time(NULL));

    for(int i = 0; i < size; i++){
        float r = rand() % 64;
        a[i] = r;
    }
}

void callCublas(int size){
    hipblasHandle_t h;
    hipblasCreate(&h);

    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    hipblasSdot(h, size, d_a, 1, d_b, 1, d_res);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(h);

    printf("Calling Cublas took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for(int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveGlobalMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);
    
    printf("Threads = %d, size = %d\n", threads, size );
    // Every thread can add two numbers, thus we can add 256 number in one iteration
    int bSize = size / (2 * threads);

    float n = size - 1.0;
    float expectedRes = (pow(n, 2.0) + n) / 2;          // This is the expected result for the gaussian sum

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // Random Vector initializsation:
    // vecInit (a, size);
    // vecInit (b, size);

    // vecInitGauss(a, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // to support a size bigger than 32 we need to
    // basically do another addition for each one of the blocks!

    //LOOOOOOOOL this only applies to standard warp reduction, not using global or shared Memory!!

    // Every thread can add two numbers, thus we can add 2*threads many numbers in one iteration
    naiveGlobalMem <<<bSize, threads>>> (d_a, d_b, d_res, threads * 2);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Naive with Global Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);
    
    // Just a sanity check such that we get a message, if the result is incorrect
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

void callNaiveSharedMem(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    int bSize = size / (2 * threads);
    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitGauss(a, size);

    // vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // to support a size bigger than 32 we need to
    // basically do another addition for each one of the blocks!
    //again. that is true for the reduction version only!
    naiveSharedMem<<<bSize, 128>>> (d_a, d_b, d_res, size);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Naive with Shared Memory took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for(int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    float n = size - 1.0;
    float expectedRes = (pow(n, 2.0) + n) / 2;
    float diff = res[0] - (float)size;

    if (diff > 1.0 || diff < -1.0){
        printf("The result is %f, but should be %f, difference is %f \n", res[0], expectedRes, diff);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
}

void callNaiveWarpRed(int size, int threads){
    hipEvent_t start, stop;
    float time;

    hipEventCreate (&start);
    hipEventCreate(&stop);

    float * a = (float*) malloc (sizeof(float)*size);
    float * b = (float*) malloc (sizeof(float)*size);
    float * res = (float*) malloc (sizeof(float)*size);

    float* d_a;
    float* d_b;
    float* d_res;

    hipMalloc(&d_a, sizeof(float)*size);
    hipMalloc(&d_b, sizeof(float)*size);
    hipMalloc(&d_res, sizeof(float)*size);

    // vecInit (a, size);
    // vecInit (b, size);

    vecInitOnes(a, size);
    vecInitOnes(b, size);

    hipMemcpy(d_a, a, sizeof(float)*size , hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*size , hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    naiveWarpRed<<<1,size>>> (d_a, d_b, d_res, size);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("WarpReduction Version took %fms\n", time);

    hipMemcpy(res, d_res, sizeof(float)*size, hipMemcpyDeviceToHost);

    // for (int i = 0; i< size; i++){
    //     printf("res[%d] = %f\n", i, res[i]);
    // }
    
    if (res[0] != (float)size){
        printf("The result is %f, but should be %f \n", res[0], (float) size);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
}

int main(){

    int threads = 1 << 5;
    int size = 1 << 6;
    //  for (int i = 0; i < 5; i++){
    // }
    
    callCublas(size);
    callNaiveGlobalMem(size, threads);
    callNaiveSharedMem(size, threads);
    callNaiveWarpRed(size, threads); 

    return 0;
}
